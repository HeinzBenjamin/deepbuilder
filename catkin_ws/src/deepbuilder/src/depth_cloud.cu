#include <stdio.h>
#include <hip/hip_runtime.h>

#include <string>
#include <iostream>

float *d_fields = NULL;
float *h_fields = NULL;
float *d_zvals = NULL;
float *h_zvals = NULL;

float *d_fixedXs = NULL;
float *d_fixedYs = NULL;

int res_ = 0;
int fSX_ = 0; //fieldSizeX
int fSY_ = 0; //fieldSizeY

__device__ __inline__ void min_id(float &x, const float y, int &idx, const int idy)
{
    if (y < x)
    {
        x = y;
        idx = idy;
    }
}

__global__ void find_closest(
    const int fsx, const int fsy, const int res, const float *fields,
    const float *fixedXs, const float *fixedYs,
    float *o_zvals, const float correction)
{
    __shared__ float dists[1024];
    __shared__ int ids[1024];

    float *dist_ptr = &dists[threadIdx.x];
    dist_ptr[0] = 3.4e+38f;

    int *id_ptr = &ids[threadIdx.x];
    id_ptr[0] = threadIdx.x;

    //get square distance
    const int ggid = blockIdx.x * (fsx * fsy) + threadIdx.x;
    if (threadIdx.x < (fsx * fsy) && !isnan(fields[ggid * 4]) && !isnan(fields[ggid * 4 + 1]))
    {
        dist_ptr[0] = 0.0f;
        dist_ptr[0] += (fixedXs[blockIdx.x % res] - fields[ggid * 4]) * (fixedXs[blockIdx.x % res] - fields[ggid * 4]);
        dist_ptr[0] += (fixedYs[blockIdx.x / res] - fields[ggid * 4 + 1]) * (fixedYs[blockIdx.x / res] - fields[ggid * 4 + 1]);
    }

    __syncthreads();

    for (int bla = 512; bla > 1; bla /= 2)
    {
        if (threadIdx.x < bla)
            min_id(dist_ptr[0], dist_ptr[bla - 1], id_ptr[0], id_ptr[bla - 1]);
        __syncthreads();
    }

    float candidate = fields[4 * (blockIdx.x * (fsx * fsy) + ids[0]) + 2];
    o_zvals[blockIdx.x] = (isnan(candidate) ? 0.0f : candidate) + correction;
}

void clear_pointcloud()
{
    if (d_fields != NULL)
    {
        hipFree(d_fields);
        d_fields = NULL;
    }

    if (h_fields != NULL)
    {
        free(h_fields);
        h_fields = NULL;
    }

    if (d_zvals != NULL)
    {
        hipFree(d_zvals);
        d_zvals = NULL;
    }

    if (h_zvals != NULL)
    {
        free(h_zvals);
        h_zvals = NULL;
    }

    if (d_fixedXs != NULL)
    {
        hipFree(d_fixedXs);
        d_fixedXs = NULL;
    }

    if (d_fixedYs != NULL)
    {
        hipFree(d_fixedYs);
        d_fixedYs = NULL;
    }
}

void init_pointcloud(const int res, const int fSX, const int fSY, const float minX, const float maxX, const float minY, const float maxY)
{
    if (d_fields != NULL || h_fields != NULL)
        clear_pointcloud();

    hipDeviceReset();
    int *dev = (int *)malloc(sizeof(int));
    dev[0] = -99;
    hipGetDevice(dev);
    std::cout << "Pointcloud cuda device: " << dev[0] << std::endl;
    free(dev);
    //pcl::pointXYZ is SSE aligned and therefor consists of float[4] with the last float just being padding.
    //thus we have to use more memory than necessary, but whatevs amirite
    hipMalloc((void **)&d_fields, sizeof(float) * 4 * res * res * fSX * fSY);
    h_fields = (float *)malloc(sizeof(float) * 4 * res * res * fSX * fSY);

    hipMalloc((void **)&d_zvals, sizeof(float) * res * res);
    h_zvals = (float *)malloc(sizeof(float) * res * res);

    hipMalloc((void **)&d_fixedXs, sizeof(float) * res);
    hipMalloc((void **)&d_fixedYs, sizeof(float) * res);

    for (int i = 0; i < res; i++)
    {
        float x = minX + (maxX - minX) * (float)i / (float)(res - 1);
        float y = minY + (maxY - minY) * (float)i / (float)(res - 1);
        hipMemcpy(&d_fixedXs[i], &x, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&d_fixedYs[i], &y, sizeof(float), hipMemcpyHostToDevice);
    }
}

//call with (float*)&(transformedPointcloud->points[0])
float *make_fields(const float *flattenedTransformedPointCloud,
                   const int width, const int res,
                   const int fSX, const int fSY,
                   const int fieldStartX, const int fieldStartY,
                   const int fieldOffsetX, const int fieldOffsetY,
                   const float minX, const float maxX,
                   const float minY, const float maxY)
{
    //reset memories if sizes have changed
    if (res != res_ || fSX != fSX_ || fSY != fSY_)
    {
        clear_pointcloud();
        init_pointcloud(res, fSX, fSY, minX, maxX, minY, maxY);
        res_ = res;
        fSX_ = fSX;
        fSY_ = fSY;
    }

    int offset = 0;
    for (int iY = 0; iY < res_; iY++)
    {
        for (int iX = 0; iX < res_; iX++)
        {
            for (int iiY = 0; iiY < fSY; iiY++)
            {
                const int hf = 4 * offset;
                const int ff = 4 * (width * (fieldStartY + (fieldOffsetY * iY) + iiY) + fieldStartX + iX * fieldOffsetX);
                memcpy(&h_fields[hf], &flattenedTransformedPointCloud[ff], sizeof(float) * 4 * fSX);
                offset += fSX;
            }
        }
    }

    return h_fields;
}

//fast and amazing CUDA version
float *get_Z_values(float correction)
{
    hipMemcpy(d_fields, h_fields, sizeof(float) * 4 * fSX_ * fSY_ * res_ * res_, hipMemcpyHostToDevice);
    hipMemset(d_zvals, 0, sizeof(float) * res_ * res_);
    find_closest<<<res_ * res_, 1024>>>(fSX_, fSY_, res_, d_fields, d_fixedXs, d_fixedYs, d_zvals, correction);
    int err = hipDeviceSynchronize();
    if (err != 0)
        std::cout << "POINTCLOUD CUDA ERROR: " << err << std::endl;

    hipMemcpy(h_zvals, d_zvals, sizeof(float) * res_ * res_, hipMemcpyDeviceToHost);
    return h_zvals;
}

//slow CPU version
float *get_Z_values(int res, int fSX, int fSY, float minX, float maxX, float minY, float maxY)
{
    for (int iY = 0; iY < res_; iY++)
    {

        float y = minY + (maxY - minY) * (float)iY / (float)(res - 1);

        for (int iX = 0; iX < res_; iX++)
        {
            float x = minX + (maxX - minX) * (float)iX / (float)(res - 1);
            float dist = 9999999.9;

            for (int i = 0; i < fSX * fSY; i++)
            {
                float pixX = h_fields[((iY * res_ + iX) * (fSX * fSY) + i) * 4];
                float pixY = h_fields[((iY * res_ + iX) * (fSX * fSY) + i) * 4 + 1];
                if (dist > (x - pixX) * (x - pixX) + (y - pixY) * (y - pixY))
                {
                    h_zvals[iY * res_ + iX] = h_fields[((iY * res_ + iX) * (fSX * fSY) + i) * 4 + 2];
                    dist = (x - pixX) * (x - pixX) + (y - pixY) * (y - pixY);
                }
            }
        }
    }
    return h_zvals;
}